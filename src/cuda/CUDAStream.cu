#include "hip/hip_runtime.h"

// Copyright (c) 2015-16 Tom Deakin, Simon McIntosh-Smith,
// University of Bristol HPC
//
// For full license terms please see the LICENSE file distributed with this
// source code


#include "CUDAStream.h"

void check_error(void)
{
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    exit(err);
  }
}

template <class T>
CUDAStream<T>::CUDAStream(const int ARRAY_SIZE, const int device_index)
{

  // The array size must be divisible by TBSIZE for kernel launches
  if (ARRAY_SIZE % TBSIZE != 0)
  {
    std::stringstream ss;
    ss << "Array size must be a multiple of " << TBSIZE;
    throw std::runtime_error(ss.str());
  }

  // Set device
  int count;
  hipGetDeviceCount(&count);
  check_error();
  if (device_index >= count)
    throw std::runtime_error("Invalid device index");
  hipSetDevice(device_index);
  check_error();

  // Print out device information
  std::cout << "Using CUDA device " << getDeviceName(device_index) << std::endl;
  std::cout << "Driver: " << getDeviceDriver(device_index) << std::endl;

  array_size = ARRAY_SIZE;

  // Allocate the host array for partial sums for dot kernels
  sums = (T*)malloc(sizeof(T) * DOT_NUM_BLOCKS);

  // Check buffers fit on the device
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, 0);
  if (props.totalGlobalMem < 3*ARRAY_SIZE*sizeof(T))
    throw std::runtime_error("Device does not have enough memory for all 3 buffers");

  // Create device buffers
#if defined(MANAGED)
  hipMallocManaged(&d_a, ARRAY_SIZE*sizeof(T));
  check_error();
  hipMallocManaged(&d_b, ARRAY_SIZE*sizeof(T));
  check_error();
  hipMallocManaged(&d_c, ARRAY_SIZE*sizeof(T));
  check_error();
  hipMallocManaged(&d_sum, DOT_NUM_BLOCKS*sizeof(T));
  check_error();
#elif defined(PAGEFAULT)
  d_a = (T*)malloc(sizeof(T)*ARRAY_SIZE);
  d_b = (T*)malloc(sizeof(T)*ARRAY_SIZE);
  d_c = (T*)malloc(sizeof(T)*ARRAY_SIZE);
  d_sum = (T*)malloc(sizeof(T)*DOT_NUM_BLOCKS);
#else
  hipMalloc(&d_a, ARRAY_SIZE*sizeof(T));
  check_error();
  hipMalloc(&d_b, ARRAY_SIZE*sizeof(T));
  check_error();
  hipMalloc(&d_c, ARRAY_SIZE*sizeof(T));
  check_error();
  hipMalloc(&d_sum, DOT_NUM_BLOCKS*sizeof(T));
  check_error();

#ifdef ESTIMATE
  hipMalloc(&sa, ARRAY_SIZE*sizeof(uint32_t));
  check_error();
  hipMalloc(&sb, ARRAY_SIZE*sizeof(uint32_t));
  check_error();
  hipMalloc(&sc, ARRAY_SIZE*sizeof(uint32_t));
  check_error();
  hipMalloc(&ssum, DOT_NUM_BLOCKS*sizeof(uint32_t));
  check_error();
#endif

#endif


#ifdef ESTIMATE
  std::cout << "Estimate Version\n";
#else
  std::cout << "Original Version\n";
#endif
}


template <class T>
CUDAStream<T>::~CUDAStream()
{
  free(sums);

#if defined(PAGEFAULT)
  free(d_a);
  free(d_b);
  free(d_c);
  free(d_sum);
#else
  hipFree(d_a);
  check_error();
  hipFree(d_b);
  check_error();
  hipFree(d_c);
  check_error();
  hipFree(d_sum);
  check_error();

#ifdef ESTIMATE
  hipFree(sa);
  check_error();
  hipFree(sb);
  check_error();
  hipFree(sc);
  check_error();
  hipFree(ssum);
  check_error();
#endif

#endif
}


template <typename T>
#ifdef ESTIMATE
__global__ void init_kernel(T * a, T * b, T * c, T initA, T initB, T initC, uint32_t *sa, uint32_t *sb, uint32_t *sc)
#else
__global__ void init_kernel(T * a, T * b, T * c, T initA, T initB, T initC)
#endif
{
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  a[i] = initA;
  b[i] = initB;
  c[i] = initC;

#ifdef ESTIMATE   
  record_w(sa, i)
  record_w(sb, i)
  record_w(sc, i)
#endif
}

template <class T>
void CUDAStream<T>::init_arrays(T initA, T initB, T initC)
{
#ifdef ESTIMATE
  init_kernel<<<array_size/TBSIZE, TBSIZE>>>(d_a, d_b, d_c, initA, initB, initC, sa, sb, sc);
#else
  init_kernel<<<array_size/TBSIZE, TBSIZE>>>(d_a, d_b, d_c, initA, initB, initC);
#endif
  check_error();
  hipDeviceSynchronize();
  check_error();
}

template <class T>
void CUDAStream<T>::read_arrays(std::vector<T>& a, std::vector<T>& b, std::vector<T>& c)
{
  // Copy device memory to host
#if defined(PAGEFAULT) || defined(MANAGED)
  hipDeviceSynchronize();
  for (int i = 0; i < array_size; i++)
  {
    a[i] = d_a[i];
    b[i] = d_b[i];
    c[i] = d_c[i];
  }
#else
  hipMemcpy(a.data(), d_a, a.size()*sizeof(T), hipMemcpyDeviceToHost);
  check_error();
  hipMemcpy(b.data(), d_b, b.size()*sizeof(T), hipMemcpyDeviceToHost);
  check_error();
  hipMemcpy(c.data(), d_c, c.size()*sizeof(T), hipMemcpyDeviceToHost);
  check_error();
#endif
}


template <typename T>
#ifdef ESTIMATE
__global__ void copy_kernel(const T * a, T * c, uint32_t *sa, uint32_t *sc)
#else
__global__ void copy_kernel(const T * a, T * c)
#endif
{
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  c[i] = a[i];

#ifdef ESTIMATE
    record_r(sa, i)
    record_w(sc, i)
#endif  
}

template <class T>
void CUDAStream<T>::copy()
{
#ifdef ESTIMATE
  copy_kernel<<<array_size/TBSIZE, TBSIZE>>>(d_a, d_c, sa, sc);
#else
  copy_kernel<<<array_size/TBSIZE, TBSIZE>>>(d_a, d_c);
#endif
  check_error();
  hipDeviceSynchronize();
  check_error();
}

template <typename T>
#ifdef ESTIMATE
__global__ void mul_kernel(T * b, const T * c, uint32_t *sb, uint32_t *sc)
#else
__global__ void mul_kernel(T * b, const T * c)
#endif
{
  const T scalar = startScalar;
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  b[i] = scalar * c[i];

#ifdef ESTIMATE
    record_r(sc, i)
    record_w(sb, i)
#endif 
}

template <class T>
void CUDAStream<T>::mul()
{
#ifdef ESTIMATE
  mul_kernel<<<array_size/TBSIZE, TBSIZE>>>(d_b, d_c, sb, sc);
#else
  mul_kernel<<<array_size/TBSIZE, TBSIZE>>>(d_b, d_c);
#endif
  check_error();
  hipDeviceSynchronize();
  check_error();
}

template <typename T>
#ifdef ESTIMATE
__global__ void add_kernel(const T * a, const T * b, T * c, uint32_t *sa, uint32_t *sb, uint32_t *sc)
#else
__global__ void add_kernel(const T * a, const T * b, T * c)
#endif
{
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  c[i] = a[i] + b[i];

#ifdef ESTIMATE
    record_r(sa, i)
    record_r(sb, i)
    record_w(sc, i)
#endif
}

template <class T>
void CUDAStream<T>::add()
{
#ifdef ESTIMATE
  add_kernel<<<array_size/TBSIZE, TBSIZE>>>(d_a, d_b, d_c, sa, sb, sc);
#else  
  add_kernel<<<array_size/TBSIZE, TBSIZE>>>(d_a, d_b, d_c);
#endif
  check_error();
  hipDeviceSynchronize();
  check_error();
}

template <typename T>
#ifdef ESTIMATE
__global__ void triad_kernel(T * a, const T * b, const T * c, uint32_t *sa, uint32_t *sb, uint32_t *sc)
#else
__global__ void triad_kernel(T * a, const T * b, const T * c)
#endif
{
  const T scalar = startScalar;
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  a[i] = b[i] + scalar * c[i];

#ifdef ESTIMATE
    record_r(sb, i)
    record_r(sc, i)
    record_w(sa, i)
#endif
}

template <class T>
void CUDAStream<T>::triad()
{
#ifdef ESTIMATE
  triad_kernel<<<array_size/TBSIZE, TBSIZE>>>(d_a, d_b, d_c, sa, sb, sc);
#else
  triad_kernel<<<array_size/TBSIZE, TBSIZE>>>(d_a, d_b, d_c);
#endif
  check_error();
  hipDeviceSynchronize();
  check_error();
}

template <typename T>
#ifdef ESTIMATE
__global__ void nstream_kernel(T * a, const T * b, const T * c, uint32_t *sa, uint32_t *sb, uint32_t *sc)
#else
__global__ void nstream_kernel(T * a, const T * b, const T * c)
#endif
{
  const T scalar = startScalar;
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  a[i] += b[i] + scalar * c[i];

#ifdef ESTIMATE
    record_r(sb, i)
    record_r(sc, i)
    record_w(sa, i)
#endif
}

template <class T>
void CUDAStream<T>::nstream()
{
#ifdef ESTIMATE
  nstream_kernel<<<array_size/TBSIZE, TBSIZE>>>(d_a, d_b, d_c, sa, sb, sc);
#else
  nstream_kernel<<<array_size/TBSIZE, TBSIZE>>>(d_a, d_b, d_c);
#endif
  check_error();
  hipDeviceSynchronize();
  check_error();
}

template <class T>
#ifdef ESTIMATE
__global__ void dot_kernel(const T * a, const T * b, T * sum, int array_size, uint32_t *sa, uint32_t *sb, uint32_t *ssum)
#else
__global__ void dot_kernel(const T * a, const T * b, T * sum, int array_size)
#endif
{
  __shared__ T tb_sum[TBSIZE];

  int i = blockDim.x * blockIdx.x + threadIdx.x;
  const size_t local_i = threadIdx.x;

  tb_sum[local_i] = 0.0;
  for (; i < array_size; i += blockDim.x*gridDim.x) {
    tb_sum[local_i] += a[i] * b[i];
#ifdef ESTIMATE
    record_r(sa, i)
    record_r(sb, i)
#endif
  }

  for (int offset = blockDim.x / 2; offset > 0; offset /= 2)
  {
    __syncthreads();
    if (local_i < offset)
    {
      tb_sum[local_i] += tb_sum[local_i+offset];
    }
  }

  if (local_i == 0) {
    sum[blockIdx.x] = tb_sum[local_i];
#ifdef ESTIMATE
    record_w(ssum, blockIdx.x)
#endif
  }
}

template <class T>
T CUDAStream<T>::dot()
{
#ifdef ESTIMATE
  dot_kernel<<<DOT_NUM_BLOCKS, TBSIZE>>>(d_a, d_b, d_sum, array_size, sa, sb, ssum);
#else  
  dot_kernel<<<DOT_NUM_BLOCKS, TBSIZE>>>(d_a, d_b, d_sum, array_size);
#endif
  check_error();

#if defined(MANAGED) || defined(PAGEFAULT)
  hipDeviceSynchronize();
  check_error();
#else
  hipMemcpy(sums, d_sum, DOT_NUM_BLOCKS*sizeof(T), hipMemcpyDeviceToHost);
  check_error();
#endif

  T sum = 0.0;
  for (int i = 0; i < DOT_NUM_BLOCKS; i++)
  {
#if defined(MANAGED) || defined(PAGEFAULT)
    sum += d_sum[i];
#else
    sum += sums[i];
#endif
  }

  return sum;
}

void listDevices(void)
{
  // Get number of devices
  int count;
  hipGetDeviceCount(&count);
  check_error();

  // Print device names
  if (count == 0)
  {
    std::cerr << "No devices found." << std::endl;
  }
  else
  {
    std::cout << std::endl;
    std::cout << "Devices:" << std::endl;
    for (int i = 0; i < count; i++)
    {
      std::cout << i << ": " << getDeviceName(i) << std::endl;
    }
    std::cout << std::endl;
  }
}


std::string getDeviceName(const int device)
{
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, device);
  check_error();
  return std::string(props.name);
}


std::string getDeviceDriver(const int device)
{
  hipSetDevice(device);
  check_error();
  int driver;
  hipDriverGetVersion(&driver);
  check_error();
  return std::to_string(driver);
}

template class CUDAStream<float>;
template class CUDAStream<double>;
